// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_cc.cu
 *
 * @brief Simple test driver program for connected component.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

#include "EvqueueManager.h"

// CC includes
#include <gunrock/app/cc/cc_enactor.cuh>
#include <gunrock/app/cc/cc_problem.cuh>
#include <gunrock/app/cc/cc_functor.cuh>

// Operator includes
#include <gunrock/oprtr/filter/kernel.cuh>

// Boost includes for CPU CC reference algorithms
#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/connected_components.hpp>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::cc;


/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

template <typename VertexId>
struct CcList
{
    VertexId        root;
    unsigned int    histogram;

    CcList(VertexId root, unsigned int histogram) :
        root(root), histogram(histogram) {}
};

template<typename CcList>
bool CCCompare(
    CcList elem1,
    CcList elem2)
{
    return elem1.histogram > elem2.histogram;
}


/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        "\ntest_cc <graph type> <graph type args> [--device=<device_index>] "
        "[--instrumented] [--quick=<0|1>]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --quick If set will skip the CPU validation code. Default: 0.\n"
        );
}

/**
 * @brief Displays the CC result (i.e., number of components)
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in] comp_ids Host-side vector to store computed component id for each node
 * @param[in] nodes Number of nodes in the graph
 * @param[in] num_components Number of connected components in the graph
 * @param[in] roots Host-side vector stores the root for each node in the graph
 * @param[in] histogram Histogram of connected component ids
 */
template<typename VertexId, typename SizeT>
void DisplaySolution(
    VertexId     *comp_ids,
    SizeT        nodes,
    unsigned int num_components,
    VertexId     *roots,
    unsigned int *histogram)
{
    typedef CcList<VertexId> CcListType;
    printf("Number of Components: %d\n", num_components);

    if (nodes <= 40)
    {
        PrintFormatArray (comp_ids, nodes, "%4d", 10);
        /*
        printf("[");
        for (VertexId i = 0; i < nodes; ++i)
        {
            PrintValue(i);
            printf(":");
            PrintValue(comp_ids[i]);
            printf(",");
            printf(" ");
        }
        printf("]\n");
        */
    }
    else
    {
        //sort the components by size
        CcListType *cclist =
            (CcListType*)malloc(sizeof(CcListType) * num_components);
        for (int i = 0; i < num_components; ++i)
        {
            cclist[i].root = roots[i];
            cclist[i].histogram = histogram[i];
        }
        std::stable_sort(
            cclist, cclist + num_components, CCCompare<CcListType>);

        // Print out at most top 10 largest components
        int top = (num_components < 10) ? num_components : 10;
        printf("Top %d largest components:\n", top);
        for (int i = 0; i < top; ++i)
        {
            printf("CC ID: %d, CC Root: %d, CC Size: %d\n",
                   i, cclist[i].root, cclist[i].histogram);
        }

        free(cclist);
    }
}

/**
 * Performance/Evaluation statistics
 */

/******************************************************************************
 * CC Testing Routines
 *****************************************************************************/

/**
 * @brief CPU-based reference CC algorithm using Boost Graph Library
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in] row_offsets Host-side vector stores row offsets for each node in the graph
 * @param[in] column_indices Host-side vector stores column indices for each edge in the graph
 * @param[in] num_nodes
 * @param[out] labels Host-side vector to store the component id for each node in the graph
 *
 * \return Number of connected components in the graph
 */
template<typename VertexId, typename SizeT>
unsigned int RefCPUCC(
    SizeT *row_offsets, VertexId *column_indices, int num_nodes, int *labels)
{
    using namespace boost;
    typedef adjacency_list <vecS, vecS, undirectedS> Graph;
    Graph G;
    for (int i = 0; i < num_nodes; ++i)
    {
        for (int j = row_offsets[i]; j < row_offsets[i+1]; ++j)
        {
            add_edge(i, column_indices[j], G);
        }
    }
    CpuTimer cpu_timer;
    cpu_timer.Start();
    int num_components = connected_components(G, &labels[0]);
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    printf("CPU CC finished in %lf msec.\n", elapsed);
    return num_components;
}

/**
 * @brief Run tests for connected component algorithm
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy for CC kernels
 * @param[in] iterations Number of iterations for running the test
 * @param[in] num_gpus Number of GPUs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    int max_grid_size,
    int iterations,
    int num_gpus)
{
    typedef CCProblem<
        VertexId,
        SizeT,
        Value,
        true> Problem; //use double buffer for edgemap and vertexmap.

    // Allocate host-side label array (for both reference and gpu-computed results)
    VertexId     *reference_component_ids        = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    VertexId     *h_component_ids                = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    VertexId     *reference_check                = (g_quick) ? NULL : reference_component_ids;
    unsigned int ref_num_components             = 0;

    // Allocate CC enactor map
    CCEnactor<INSTRUMENT> cc_enactor(g_verbose);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;
    util::GRError(csr_problem->Init(
                      g_stream_from_host,
                      graph,
                      num_gpus),
                  "CC Problem Initialization Failed", __FILE__, __LINE__);

    //
    // Compute reference CPU CC
    //
    if (reference_check != NULL && !g_quick)
    {
        printf("Computing reference value ...\n");
        ref_num_components = RefCPUCC(
            graph.row_offsets,
            graph.column_indices,
            graph.nodes,
            reference_check);
        printf("\n");
    }

    long long total_queued = 0;
    VertexId  num_iter = 0;
    double    avg_duty = 0.0;

    // Perform CC
    GpuTimer gpu_timer;

    float elapsed = 0.0f;

    iterations = 5000;

    struct timeval start, end;
    for (int iter = 0; iter < iterations; ++iter)
    {
        util::GRError(
            csr_problem->Reset(cc_enactor.GetFrontierType()),
            "CC Problem Data Reset Failed", __FILE__, __LINE__);

        gpu_timer.Start();
        if(iter%/*50*/7==0)
        {
        gettimeofday(&start, NULL);
        }
        util::GRError(
            cc_enactor.template Enact<Problem>(csr_problem, max_grid_size),
            "CC Problem Enact Failed", __FILE__, __LINE__);
        if(iter%/*50*/7==/*49*/6)
        {
        gettimeofday(&end, NULL);
        std::cerr << "[CC] ---- " << (end.tv_sec - start.tv_sec)*1000000+(end.tv_usec - start.tv_usec) << std::endl;
        }
        EvqueueSynch();
        gpu_timer.Stop();

        elapsed += gpu_timer.ElapsedMillis();
        //printf("iteration %d, time: %.5f\n", iter+1, gpu_timer.ElapsedMillis());
    }
    elapsed /= iterations;

    cc_enactor.GetStatistics(total_queued, num_iter, avg_duty);

    // Copy out results
    util::GRError(
        csr_problem->Extract(h_component_ids),
        "CC Problem Data Extraction Failed", __FILE__, __LINE__);

    // Validity
    if (!g_quick)
    {
        if (ref_num_components == csr_problem->num_components)
            printf("CORRECT.\n");
        else
            printf("INCORRECT. Ref Component Count: %d,"
                   "GPU Computed Component Count: %d\n",
                   ref_num_components, csr_problem->num_components);
    }

    // Compute size and root of each component
    VertexId     *h_roots      = new VertexId[csr_problem->num_components];
    unsigned int *h_histograms = new unsigned int[csr_problem->num_components];

    csr_problem->ComputeCCHistogram(h_component_ids, h_roots, h_histograms);

    // Display Solution
    DisplaySolution(h_component_ids, graph.nodes,
                    csr_problem->num_components,
                    h_roots, h_histograms);

    if (h_roots) delete[] h_roots;
    if (h_histograms) delete[] h_histograms;

    printf("[GPU Connected Component] finished.\n");
    printf(" elapsed: %.4f ms\n", elapsed);
    printf(" num_iterations: %d\n", num_iter);

    // Cleanup
    if (csr_problem) delete csr_problem;
    if (reference_component_ids) free(reference_component_ids);
    if (h_component_ids) free(h_component_ids);

    hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args)
{
    bool instrumented  = false; // Whether or not to collect instrumentation from kernels
    int  max_grid_size = 0;     // Maximum grid size (0: leave it up to the enactor)
    int  num_gpus      = 1;     // Number of GPUs for multi-gpu enactor to use
    int  iterations    = 1;     // Default run test times
    g_quick            = 1;     // Whether or not to skip ref validation

    instrumented = args.CheckCmdLineFlag("instrumented");
    g_verbose    = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("quick", g_quick);
    args.GetCmdLineArgument("iteration-num", iterations);

    if (instrumented)
    {
        RunTests<VertexId, Value, SizeT, true>(
            graph,
            max_grid_size,
            iterations,
            num_gpus);
    }
    else
    {
        RunTests<VertexId, Value, SizeT, false>(
            graph,
            max_grid_size,
            iterations,
            num_gpus);
    }
}

/******************************************************************************
 * Main
 ******************************************************************************/

int main( int argc, char** argv)
{
    EvqueueCreate(4);
    CommandLineArgs args(argc, argv);

    if ((argc < 2) || (args.CheckCmdLineFlag("help")))
    {
        Usage();
        return 1;
    }

    DeviceInit(args);
    hipSetDeviceFlags(hipDeviceMapHost);

    // Parse graph-contruction params
    g_undirected = false; //Does not make undirected graph

    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1)
    {
        Usage();
        return 1;
    }

    //
    // Construct graph and perform search(es)
    //

    if (graph_type == "market")
    {
        // Matrix-market coordinate-formatted graph file

        typedef int VertexId;                   // Use as the node identifier
        typedef int Value;                      // Use as the value type
        typedef int SizeT;                      // Use as the graph size type
        Csr<VertexId, Value, SizeT> csr(false); // default for stream_from_host

        if (graph_args < 1) { Usage(); return 1; }

        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<false>(
                market_filename,
                csr,
                g_undirected,
                false) != 0) // no inverse graph
        {
            return 1;
        }

        csr.PrintHistogram();
        fflush(stdout);

        // Run tests
        RunTests(csr, args);
    }
    else
    {
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }
    EvqueueDestroy();
    return 0;
}
