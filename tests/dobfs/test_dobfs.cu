// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

#include "EvqueueManager.h"

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// DOBFS includes
#include <gunrock/app/dobfs/dobfs_enactor.cuh>
#include <gunrock/app/dobfs/dobfs_problem.cuh>
#include <gunrock/app/dobfs/dobfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;
using namespace gunrock::app::dobfs;


/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;
float g_alpha;
float g_beta;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf (
        " test_dobfs <graph type> <graph type args> [--device=<device_index>]\n"
        " [--src=<source_index>] [--instrumented] [--idempotence=<0|1>] [--v]\n"
        " [--undirected] [--iteration-num=<num>] [--quick=<0|1>] [--mark-pred]\n"
        " [--queue-sizing=<scale factor>]\n"
        "\n"
        "Graph types and args:\n"
        "  market <file>\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed / undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>   Set GPU device for running the test. [Default: 0].\n"
        "  --undirected              Treat the graph as undirected (symmetric).\n"
        "  --idempotence=<0 or 1>    Enable: 1, Disable: 0 [Default: Enable].\n"
        "  --instrumented            Keep kernels statics [Default: Disable].\n"
        "                            total_queued, search_depth and barrier duty\n"
        "                            (a relative indicator of load imbalance.)\n"
        "  --src=<source vertex id>  Begins BFS from the source [Default: 0].\n"
        "                            If randomize: from a random source vertex.\n"
        "                            If largestdegree: from largest degree vertex.\n"
        "  --quick=<0 or 1>          Skip the CPU validation: 1, or not: 0 [Default: 1].\n"
        "  --mark-pred               Keep both label info and predecessor info.\n"
        "  --queue-sizing=<factor>   Allocates a frontier queue sized at: \n"
        "                            (graph-edges * <scale factor>). [Default: 1.0]\n"
        "  --v                       Print verbose per iteration debug info.\n"
        "  --iteration-num=<number>  Number of runs to perform the test [Default: 1].\n"
        );
}

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] preds Predecessor node id for each node.
 * @param[in] nodes Number of nodes in the graph.
 * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
 * @param[in] ENABLE_IDEMPOTENCE Whether to enable idempotence mode.
 */
template<typename VertexId, typename SizeT>
void DisplaySolution (VertexId *source_path,
                      VertexId *preds,
                      SizeT nodes,
                      bool MARK_PREDECESSORS,
                      bool ENABLE_IDEMPOTENCE)
{
    if (nodes > 40) nodes = 40;
    printf("\nFirst %d labels of the GPU result.\n", nodes);
    PrintFormatArray (source_path, nodes, "%4d", 10);
    if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE)
    {
        printf("\n First %d predecessors:\n", nodes);
        PrintFormatArray (preds, nodes, "%8d", 5);
    }
    /*
    printf("[");
    for (VertexId i = 0; i < nodes; ++i) {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE) {
            printf(",");
            PrintValue(preds[i]);
        }
        printf(" ");
    }
    printf("]\n");
    */
}

/**
 * Performance/Evaluation statistics
 */

struct Stats {
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where BFS starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the BFS algorithm
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    bool MARK_PREDECESSORS,
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    VertexId            *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] > -1) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0) {
        redundant_work = ((double) total_queued - edges_visited) / edges_visited;
        // measure duplicate edges put through queue
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5) {
        printf("Fewer than 5 vertices visited.\n");
    } else {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf("\n elapsed: %.4f ms, rate: %.4f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0) printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0) {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges_visited: %lld",
               (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0) {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0) {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }

}




/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference BFS ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] source_path Host-side vector to store CPU computed labels for each node
 * @param[in] src Source node where BFS starts
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void SimpleReferenceBfs(
    const Csr<VertexId, Value, SizeT>       &graph,
    VertexId                                *source_path,
    VertexId                                src)
{
    // Initialize distances
    for (VertexId i = 0; i < graph.nodes; ++i)
    {
        source_path[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    //Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty())
    {

        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        int edges_begin = graph.row_offsets[dequeued_node];
        int edges_end = graph.row_offsets[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge)
        {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph.column_indices[edge];
            if (source_path[neighbor] == -1)
            {
                source_path[neighbor] = neighbor_dist;
                if (search_depth < neighbor_dist)
                {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. Search depth is: %d\n",
           elapsed, search_depth);
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 * @tparam ENABLE_IDEMPOTENCE
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] inv_graph Reference to the inverse CSC graph we process on
 * @param[in] src Source node where BFS starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 * @param[in] alpha Tuning parameter for switching to reverse bfs
 * @param[in] beta Tuning parameter for switching back to normal bfs
 * @param[in] iterations Number of iterations for running the test
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    const Csr<VertexId, Value, SizeT> &inv_graph,
    VertexId src,
    int max_grid_size,
    int num_gpus,
    double max_queue_sizing,
    float alpha,        // Tuning parameter for switching to reverse bfs
    float beta,         // Tuning parameter for switching back to normal bfs
    int iterations,
    CudaContext& context)
{
    typedef DOBFSProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS,
        ENABLE_IDEMPOTENCE,
        (MARK_PREDECESSORS && ENABLE_IDEMPOTENCE)> Problem; // does not use double buffer

    // Allocate host-side label array (for both reference and gpu-computed results)
    VertexId    *reference_labels       = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    VertexId    *h_labels               = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    VertexId    *reference_check        = (g_quick) ? NULL : reference_labels;
    VertexId    *h_preds                = NULL;
    if (MARK_PREDECESSORS)
    {
        h_preds = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    }


    // Allocate BFS enactor map
    DOBFSEnactor<INSTRUMENT> dobfs_enactor(g_verbose);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;

    util::GRError(csr_problem->Init(
                      g_stream_from_host,
                      g_undirected,
                      graph,
                      inv_graph,
                      num_gpus,
                      alpha,
                      beta),
                  "Problem DOBFS Initialization Failed", __FILE__, __LINE__);

    //
    // Compute reference CPU BFS solution for source-distance
    //
    if (reference_check != NULL)
    {
        printf("Computing reference value ...\n");
        SimpleReferenceBfs(
            graph,
            reference_check,
            src);
        printf("\n");
    }

    Stats *stats = new Stats("GPU DOBFS");

    long long           total_queued = 0;
    VertexId            search_depth = 0;
    double              avg_duty = 0.0;

    // Perform BFS
    GpuTimer gpu_timer;

    float elapsed = 0.0f;
    iterations = 100;
    struct timeval start, end;
    for (int iter=0; iter < iterations; ++iter)
    {
        std::cout << "Iteration " << iter << std::endl;
        util::GRError(
            csr_problem->Reset(
                src, dobfs_enactor.GetFrontierType(), max_queue_sizing),
            "DOBFS Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        util::GRError(
            dobfs_enactor.template Enact<Problem>(
                context, csr_problem, src, max_grid_size),
            "DOBFS Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();
        gettimeofday(&end, NULL);
        std::cerr << "[HITS] ---- " << (end.tv_sec - start.tv_sec)*1000000+(end.tv_usec - start.tv_usec) << std::endl;
        elapsed += gpu_timer.ElapsedMillis();
	EvqueueSynch();
    }
    elapsed /= iterations;

    dobfs_enactor.GetStatistics(total_queued, search_depth, avg_duty);

    // Copy out results
    util::GRError(
        csr_problem->Extract(h_labels, h_preds),
        "DOBFS Problem Data Extraction Failed", __FILE__, __LINE__);

    // Verify the result
    if (reference_check != NULL)
    {
        if (!MARK_PREDECESSORS)
        {
            printf("Validity: ");
            CompareResults(h_labels, reference_check, graph.nodes, true);
        }
    }

    // Display Solution
    DisplaySolution(
        h_labels, h_preds, graph.nodes, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE);

    DisplayStats<MARK_PREDECESSORS>(
        *stats,
        src,
        h_labels,
        graph,
        elapsed,
        search_depth,
        total_queued,
        avg_duty);

    // Cleanup
    delete stats;
    if (csr_problem) delete csr_problem;
    if (reference_labels) free(reference_labels);
    if (h_labels) free(h_labels);
    if (h_preds) free(h_preds);

    hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] inv_graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    Csr<VertexId, Value, SizeT> &inv_graph,
    CommandLineArgs &args,
    CudaContext& context)
{
    VertexId    src              = -1;  // Use whatever the specified graph-type's default is
    std::string src_str;
    bool        instrumented     = 0;   // Whether or not to collect instrumentation from kernels
    bool        mark_pred        = 0;   // Whether or not to mark src-distance vs. parent vertices
    bool        idempotence      = 1;   // Whether or not to enable idempotence operation
    int         max_grid_size    = 0;   // maximum grid size (0: leave it up to the enactor)
    int         num_gpus         = 1;   // Number of GPUs for multi-gpu enactor to use
    double      max_queue_sizing = 1.0; // Maximum size scaling factor for work queues (e.g., 1.0 creates n and m-element vertex and edge frontiers).
    int         iterations       = 1;   // Number of runs
    g_quick                      = 1;   // Whether or not to skip ref validation

    instrumented = args.CheckCmdLineFlag("instrumented");
    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty())
    {
        src = 0;
    }
    else if (src_str.compare("randomize") == 0)
    {
        src = graphio::RandomNode(graph.nodes);
    }
    else if (src_str.compare("largestdegree") == 0)
    {
        int temp;
        src = graph.GetNodeWithHighestDegree(temp);
    }
    else
    {
        args.GetCmdLineArgument("src", src);
    }

    mark_pred = args.CheckCmdLineFlag("mark-pred");
    g_verbose = args.CheckCmdLineFlag("v");

    args.GetCmdLineArgument("iteration-num", iterations);
    args.GetCmdLineArgument("grid-size", max_grid_size);
    args.GetCmdLineArgument("idempotence", idempotence);
    args.GetCmdLineArgument("queue-sizing", max_queue_sizing);
    args.GetCmdLineArgument("quick", g_quick);
    args.GetCmdLineArgument("alpha", g_alpha);
    args.GetCmdLineArgument("beta", g_beta);

    if (g_alpha == 0.0f) g_alpha = 12.0f;
    if (g_beta == 0.0f)  g_beta  = 6.0f;

    // printf("alpha: %5f, beta: %5f\n", g_alpha, g_beta);

    if (instrumented)
    {
        if (mark_pred)
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, true, true, true>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, true, true, false>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
        }
        else
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, true, false, true>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, true, false, false>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
        }
    }
    else
    {
        if (mark_pred)
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, false, true, true>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, false, true, false>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
        }
        else
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, false, false, true>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, false, false, false>(
                    graph,
                    inv_graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    g_alpha,
                    g_beta,
                    iterations,
                    context);
            }
        }
    }
}

/******************************************************************************
 * Main
 ******************************************************************************/
int main( int argc, char** argv)
{
    EvqueueCreate(2);

    CommandLineArgs args(argc, argv);

    if ((argc < 2) || (args.CheckCmdLineFlag("help")))
    {
        Usage();
        return 1;
    }

    //DeviceInit(args);
    //hipSetDeviceFlags(hipDeviceMapHost);

    int dev = 0;
    args.GetCmdLineArgument("device", dev);
    ContextPtr context = mgpu::CreateCudaDevice(dev);

    //srand(0); // Presently deterministic
    //srand(time(NULL));

    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");

    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1)
    {
        Usage();
        return 1;
    }

    //
    // Construct graph and perform search(es)
    //

    if (graph_type == "market")
    {
        // Matrix-market coordinate-formatted graph file

        typedef int VertexId;                   // Use as the node identifier
        typedef int Value;                      // Use as the value type
        typedef int SizeT;                      // Use as the graph size type
        Csr<VertexId, Value, SizeT> csr(false); // default for stream_from_host

        Csr<VertexId, Value, SizeT> inv_csr(false);

        if (graph_args < 1) { Usage(); return 1; }

        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<false>(
                market_filename,
                csr,
                g_undirected,
                false) != 0)
        {
            return 1;
        }

        if (!g_undirected)
        {
            if (graphio::BuildMarketGraph<false>(
                    market_filename,
                    inv_csr,
                    g_undirected,
                    true) != 0)
            {
                return 1;
            }
        }

        csr.PrintHistogram();

        if (!g_undirected)
        {
            // Run tests
            RunTests(csr, inv_csr, args, *context);
        }
        else
        {
            RunTests(csr, csr, args, *context);
        }

    }
    else
    {
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }
  EvqueueDestroy();
    return 0;
}
