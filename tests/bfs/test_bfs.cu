// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>
#include <gunrock/graphio/rmat.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

#include "EvqueueManager.h"

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        " test_bfs <graph type> <graph type args> [--device=<device_index>]\n"
        " [--undirected] [--src=<source_index>] [--idempotence=<0|1>] [--v]\n"
        " [--instrumented] [--iteration-num=<num>] [--traversal-mode=<0|1>]\n"
        " [--quick=<0|1>] [--mark-pred] [--queue-sizing=<scale factor>] "
        "\n"
        "Graph types and args:\n"
        "  market <file>\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed / undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>   Set GPU device for running the test. [Default: 0].\n"
        "  --undirected              Treat the graph as undirected (symmetric).\n"
        "  --idempotence=<0 or 1>    Enable: 1, Disable: 0 [Default: Enable].\n"
        "  --instrumented            Keep kernels statics [Default: Disable].\n"
        "                            total_queued, search_depth and barrier duty\n"
        "                            (a relative indicator of load imbalance.)\n"
        "  --src=<source vertex id>  Begins BFS from the source [Default: 0].\n"
        "                            If randomize: from a random source vertex.\n"
        "                            If largestdegree: from largest degree vertex.\n"
        "  --quick=<0 or 1>          Skip the CPU validation: 1, or not: 0 [Default: 1].\n"
        "  --mark-pred               Keep both label info and predecessor info.\n"
        "  --queue-sizing=<factor>   Allocates a frontier queue sized at: \n"
        "                            (graph-edges * <scale factor>). [Default: 1.0]\n"
        "  --v                       Print verbose per iteration debug info.\n"
        "  --iteration-num=<number>  Number of runs to perform the test [Default: 1].\n"
        "  --traversal-mode=<0 or 1> Set traversal strategy, 0 for Load-Balanced, \n"
        "                            1 for Dynamic-Cooperative [Default: dynamic\n"
        "                            determine based on average degree].\n"
        );
}

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] preds Predecessor node id for each node.
 * @param[in] nodes Number of nodes in the graph.
 * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
 * @param[in] ENABLE_IDEMPOTENCE Whether to enable idempotence mode.
 */
template<typename VertexId, typename SizeT>
void DisplaySolution(
    VertexId *labels,
    VertexId *preds,
    SizeT     num_nodes,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE)
{
    if (num_nodes > 40) num_nodes = 40;

    printf("\nFirst %d labels of the GPU result:\n", num_nodes);
    PrintFormatArray (labels, num_nodes, "%4d", 10);
    if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE)
    {
        printf("\nFirst %d predecessors:\n", num_nodes);
        PrintFormatArray (preds, num_nodes, "%4d", 10);
    }

    /*
    printf("[");
    for (VertexId i = 0; i < num_nodes; ++i)
    {
        PrintValue(i);
        printf(":");
        PrintValue(labels[i]);
        if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE)
        {
            printf(",");
            PrintValue(preds[i]);
        }
        printf(" ");
    }
    printf("]\n");
    */
}

/**
 * Performance/Evaluation statistics
 */
struct Stats
{
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) :
        name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where BFS starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the BFS algorithm
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    bool MARK_PREDECESSORS,
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    VertexId            *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i)
    {
        if (h_labels[i] > -1)
        {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0)
    {
        // measure duplicate edges put through queue
        redundant_work = ((double)total_queued - edges_visited) / edges_visited;
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5)
    {
        printf("Fewer than 5 vertices visited.\n");
    }
    else
    {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf("\n elapsed: %.4f ms, rate: %.4f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0)
            printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0)
        {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges_visited: %lld",
               (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0)
        {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0)
        {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
}

/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference BFS ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] source_path Host-side vector to store CPU computed labels for each node
 * @param[in] predecessor Host-side vector to store CPU computed predecessor for each node
 * @param[in] src Source node where BFS starts
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT,
    bool MARK_PREDECESSORS>
void SimpleReferenceBfs(
    const Csr<VertexId, Value, SizeT>       &graph,
    VertexId                                *source_path,
    VertexId                                *predecessor,
    VertexId                                src)
{
    // Initialize distances
    for (VertexId i = 0; i < graph.nodes; ++i)
    {
        source_path[i] = -1;
        if (MARK_PREDECESSORS)
            predecessor[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    // Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty())
    {
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        int edges_begin = graph.row_offsets[dequeued_node];
        int edges_end = graph.row_offsets[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge)
        {
            // Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph.column_indices[edge];
            if (source_path[neighbor] == -1)
            {
                source_path[neighbor] = neighbor_dist;
                if (MARK_PREDECESSORS)
                    predecessor[neighbor] = dequeued_node;
                if (search_depth < neighbor_dist)
                {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    if (MARK_PREDECESSORS)
        predecessor[src] = -1;

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. cpu_search_depth: %d\n",
           elapsed, search_depth);
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where BFS starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 * @param[in] iterations Number of iterations for running the test
 * @param[in] traversal_mode Graph traversal mode: Load-balanced or Dynamic cooperative
 * @param[in] context CudaContext pointer for moderngpu APIs
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    VertexId src,
    int max_grid_size,
    int num_gpus,
    double max_queue_sizing,
    int iterations,
    int traversal_mode,
    CudaContext& context)
{
    typedef BFSProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS,
        ENABLE_IDEMPOTENCE,
        (MARK_PREDECESSORS && ENABLE_IDEMPOTENCE)> Problem; // does not use double buffer

    // Allocate host-side label array (for both reference and gpu-computed results)
    VertexId    *reference_labels       = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    VertexId    *reference_preds        = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    VertexId    *h_labels               = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
    VertexId    *reference_check_label  = (g_quick) ? NULL : reference_labels;
    VertexId    *reference_check_preds  = NULL;
    VertexId    *h_preds                = NULL;
    if (MARK_PREDECESSORS)
    {
        h_preds = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
        if (!g_quick)
        {
            reference_check_preds = reference_preds;
        }
    }

    // Allocate BFS enactor map
    BFSEnactor<INSTRUMENT> bfs_enactor(g_verbose);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;
    util::GRError(csr_problem->Init(
                      g_stream_from_host,
                      graph,
                      num_gpus),
                  "Problem BFS Initialization Failed", __FILE__, __LINE__);

    Stats *stats = new Stats("GPU BFS");

    long long           total_queued = 0;
    VertexId            search_depth = 0;
    double              avg_duty = 0.0;

    // Perform BFS
    GpuTimer gpu_timer;

    float elapsed = 0.0f;

    iterations = 100000;
    struct timeval start, end;
    for (int iter = 0; iter < iterations; ++iter)
    {
        std::cerr << "Iteration " << iter << std::endl;
        util::GRError(
            csr_problem->Reset(src, bfs_enactor.GetFrontierType(),
                               max_queue_sizing),
            "BFS Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        if(iter%100==0)
        {
        gettimeofday(&start, NULL);
        }
        util::GRError(
            bfs_enactor.template Enact<Problem>(context, csr_problem, src,
                                                max_grid_size, traversal_mode),
            "BFS Problem Enact Failed", __FILE__, __LINE__);
        if(iter%100==99)
        {
        gettimeofday(&end, NULL);
        std::cerr << "[BFS] ---- " << (end.tv_sec - start.tv_sec)*1000000+(end.tv_usec - start.tv_usec) << std::endl;
        }
        gpu_timer.Stop();

        elapsed += gpu_timer.ElapsedMillis();
        //EvqueueSynch();
    }

    elapsed /= iterations;

    bfs_enactor.GetStatistics(total_queued, search_depth, avg_duty);

    // Copy out results
    util::GRError(
        csr_problem->Extract(h_labels, h_preds),
        "BFS Problem Data Extraction Failed", __FILE__, __LINE__);

    //
    // Compute reference CPU BFS solution for source-distance
    //
    if (reference_check_label != NULL)
    {
        printf("Computing reference value ...\n");
        SimpleReferenceBfs<VertexId, Value, SizeT, MARK_PREDECESSORS>(
            graph,
            reference_check_label,
            reference_check_preds,
            src);
        printf("\n");
    }

    // Verify the result
    if (reference_check_label != NULL)
    {
        if (!ENABLE_IDEMPOTENCE)
        {
            printf("Label Validity: ");
            int error_num = CompareResults(
                h_labels, reference_check_label, graph.nodes, true);
            if (error_num > 0)
                printf("%d errors occurred.\n", error_num);
        }
        else
        {
            if (!MARK_PREDECESSORS)
            {
                printf("Label Validity: ");
                int error_num = CompareResults(
                    h_labels, reference_check_label, graph.nodes, true);
                if (error_num > 0)
                    printf("%d errors occurred.\n", error_num);
            }
        }
    }

    // Display Solution
    DisplaySolution(
        h_labels, h_preds, graph.nodes, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE);

    DisplayStats<MARK_PREDECESSORS>(
        *stats,
        src,
        h_labels,
        graph,
        elapsed,
        search_depth,
        total_queued,
        avg_duty);

    // Cleanup
    delete stats;
    if (csr_problem) delete csr_problem;
    if (reference_labels) free(reference_labels);
    if (h_labels) free(h_labels);
    if (h_preds) free(h_preds);

    hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args,
    CudaContext& context)
{
    VertexId    src              = -1;  // Use whatever the specified graph-type's default is
    std::string src_str;
    bool        instrumented     = 0;   // Whether or not to collect instrumentation from kernels
    bool        mark_pred        = 0;   // Whether or not to mark src-distance vs. parent vertices
    bool        idempotence      = 1;   // Whether or not to enable idempotence operation
    int         max_grid_size    = 0;   // Maximum grid size (0: leave it up to the enactor)
    int         num_gpus         = 1;   // Number of GPUs for multi-gpu enactor to use
    double      max_queue_sizing = 1.0; // Maximum size scaling factor for work queues (e.g., 1.0 creates n and m-element vertex and edge frontiers).
    int         iterations       = 1;   // Number of runs for testing
    int         traversal_mode   = -1;  // Load-balacned or Dynamic cooperative
    g_quick                      = 1;   // Whether or not to skip reference validation
    // source vertex
    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty())
    {
        src = 0;
    }
    else if (src_str.compare("randomize") == 0)
    {
        src = graphio::RandomNode(graph.nodes);
    }
    else if (src_str.compare("largestdegree") == 0)
    {
        int max_degree;
        src = graph.GetNodeWithHighestDegree(max_degree);
        printf("Using highest degree (%d) vertex: %d\n", max_degree, src);
    }
    else
    {
        args.GetCmdLineArgument("src", src);
    }

    // traversal mode
    args.GetCmdLineArgument("traversal-mode", traversal_mode);
    if (traversal_mode == -1)
    {
        traversal_mode = graph.GetAverageDegree() > 8 ? 0 : 1;
    }

    // printf("Display neighbor list of src:\n");
    // graph.DisplayNeighborList(src);

    mark_pred    = args.CheckCmdLineFlag("mark-pred");
    g_verbose    = args.CheckCmdLineFlag("v");
    instrumented = args.CheckCmdLineFlag("instrumented");

    args.GetCmdLineArgument("quick", g_quick);
    args.GetCmdLineArgument("iteration-num", iterations);
    args.GetCmdLineArgument("grid-size", max_grid_size);
    args.GetCmdLineArgument("idempotence", idempotence);
    args.GetCmdLineArgument("queue-sizing", max_queue_sizing);

    if (instrumented)
    {
        if (mark_pred)
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, true, true, true>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, true, true, false>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
        }
        else
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, true, false, true>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, true, false, false>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
        }
    }
    else
    {
        if (mark_pred)
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, false, true, true>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, false, true, false>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
        }
        else
        {
            if (idempotence)
            {
                RunTests<VertexId, Value, SizeT, false, false, true>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
            else
            {
                RunTests<VertexId, Value, SizeT, false, false, false>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    iterations,
                    traversal_mode,
                    context);
            }
        }
    }
}

/******************************************************************************
 * Main
 ******************************************************************************/
int main( int argc, char** argv)
{
    EvqueueCreate(2);

    CommandLineArgs args(argc, argv);

    if ((argc < 2) || (args.CheckCmdLineFlag("help")))
    {
        Usage();
        return 1;
    }

    //DeviceInit(args);
    //hipSetDeviceFlags(hipDeviceMapHost);

    int dev = 0;
    args.GetCmdLineArgument("device", dev);
    ContextPtr context = mgpu::CreateCudaDevice(dev);

    //srand(0); // Presently deterministic
    //srand(time(NULL));

    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");

    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1)
    {
        Usage();
        return 1;
    }

    //
    // Construct graph and perform search(es)
    //

    typedef int VertexId;                   // Use as the node identifier
    typedef int Value;                      // Use as the value type
    typedef int SizeT;                      // Use as the graph size type
    Csr<VertexId, Value, SizeT> csr(false); // default for stream_from_host
    if (graph_args < 1) { Usage(); return 1; }

    if (graph_type == "market")
    {
        // Matrix-market coordinate-formatted graph file
        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<false>(
                market_filename,
                csr,
                g_undirected,
                false) != 0) // no inverse graph
        {
            return 1;
        }

        csr.PrintHistogram();
        RunTests(csr, args, *context);
    }

    else if (graph_type == "rmat")
    {
        // parse rmat parameters
        SizeT rmat_nodes = 1 << 10;
        SizeT rmat_edges = 1 << 10;
        double rmat_a = 0.55;
        double rmat_b = 0.2;
        double rmat_c = 0.2;
        double rmat_d = 0.05;

        if (graphio::BuildRmatGraph<false>(
                rmat_nodes,
                rmat_edges,
                csr,
                g_undirected,
                rmat_a,
                rmat_b,
                rmat_c,
                rmat_d) != 0)
        {
            return 1;
        }

        csr.PrintHistogram();
        RunTests(csr, args, *context);
    }
    else
    {
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }
    EvqueueDestroy();
    return 0;
}
